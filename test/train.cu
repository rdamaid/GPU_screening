#include <iostream>

#include "../CPU/linear.h"
#include "../CPU/relu.h"
#include "../CPU/train.h"
#include "../GPU/linear.h"
#include "../GPU/relu.h"
#include "../GPU/train.h"
#include "../utils/utils.h"


int main(){
    int bs, n_in, n_hidden, n_epochs;
    int sz_inp, sz_weights1, sz_hidden;
    float *inp_cpu, *out_cpu, *inp_gpu, *out_gpu;
    
    for (int i=0; i<8; i++){
        std::cout << "Iteration " << i+1 << std::endl;

        bs = random_int(8, 64);
        n_in = random_int(16, 32);
        n_epochs = random_int(1, 4);

        n_hidden = n_in/2;
        sz_inp = bs*n_in;
        sz_weights1 = n_in*n_hidden;
        sz_hidden = bs*n_hidden;

        inp_cpu = new float[sz_inp];
        hipMallocManaged(&inp_gpu, sz_inp*sizeof(float));

        out_cpu = new float[bs];
        hipMallocManaged(&out_gpu, bs*sizeof(float));

        fill_array(inp_cpu, sz_inp);
        set_eq(inp_gpu, inp_cpu, sz_inp);
        
        fill_array(out_cpu, bs);
        set_eq(out_gpu, out_cpu, bs);

        Linear_CPU* lin1_cpu = new Linear_CPU(bs, n_in, n_hidden);
        Linear_GPU* lin1_gpu = new Linear_GPU(bs, n_in, n_hidden);
        set_eq(lin1_gpu->weights, lin1_cpu->weights, sz_weights1);
        
        ReLU_CPU* relu1_cpu = new ReLU_CPU(sz_hidden);
        ReLU_GPU* relu1_gpu = new ReLU_GPU(sz_hidden);

        Linear_CPU* lin2_cpu = new Linear_CPU(bs, n_hidden, 1);
        Linear_GPU* lin2_gpu = new Linear_GPU(bs, n_hidden, 1);
        set_eq(lin2_gpu->weights, lin2_cpu->weights, n_hidden);

        std::vector<Module*> layers_cpu = {lin1_cpu, relu1_cpu, lin2_cpu};
        std::vector<Module*> layers_gpu = {lin1_gpu, relu1_gpu, lin2_gpu};

        Sequential_CPU seq_cpu(layers_cpu);
        Sequential_GPU seq_gpu(layers_gpu);

        std::cout << "Result of train" << std::endl;
        std::cout << "CPU" << std::endl;
        train_cpu(seq_cpu, inp_cpu, out_cpu, bs, n_in, n_epochs);
        std::cout << "GPU" << std::endl;
        train_gpu(seq_gpu, inp_gpu, out_gpu, bs, n_in, n_epochs);
    
        std::cout << "*********" << std::endl;
    }

    return 0;
}

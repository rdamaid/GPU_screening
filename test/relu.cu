#include <iostream>

#include "../CPU/relu.h"
#include "../GPU/relu.h"
#include "../utils/utils.h"


int main(){
    int bs;
    float *inp_cpu, *out_cpu, *inp_gpu, *out_gpu;

    for (int i=0; i<8; i++){
        std::cout << "Iteration " << i+1 << std::endl;
        
        bs = random_int(128, 2048);

        inp_cpu = new float[bs];
        hipMallocManaged(&inp_gpu, bs*sizeof(float));

        out_cpu = new float[bs];
        hipMallocManaged(&out_gpu, bs*sizeof(float));
    
        fill_array(inp_cpu, bs);
        set_eq(inp_gpu, inp_cpu, bs);

        ReLU_CPU relu_cpu(bs);
        ReLU_GPU relu_gpu(bs);
    
        relu_cpu.forward(inp_cpu, out_cpu);
        relu_gpu.forward(inp_gpu, out_gpu);

        std::cout << "Result of the forward pass" << std::endl; 
        test_res(relu_cpu.out, relu_gpu.out, bs);

        relu_cpu.backward();
        relu_gpu.backward();

        std::cout << "Result of the backward pass" << std::endl; 
        test_res(relu_cpu.inp, relu_gpu.inp, bs);
    }

    return 0;
}

#include <iostream>

#include "../CPU/linear.h"
#include "../GPU/linear.h"
#include "../utils/utils.h"


int main(){
    int bs, n_in, n_out;
    int sz_inp, sz_weights, sz_out;
    float *inp_cpu, *out_cpu, *inp_gpu, *out_gpu;

    for (int i=0; i<8; i++){
        std::cout << "Iteration " << i+1 << std::endl;

        bs = random_int(32, 256);
        n_in = random_int(32, 64);
        n_out = random_int(1, 32);
        
        sz_inp = bs*n_in;
        sz_weights = n_in*n_out;
        sz_out = bs*n_out;

        inp_cpu = new float[sz_inp];
        hipMallocManaged(&inp_gpu, sz_inp*sizeof(float));

        out_cpu = new float[sz_out];
        hipMallocManaged(&out_gpu, sz_out*sizeof(float));
    
        fill_array(inp_cpu, sz_inp);
        set_eq(inp_gpu, inp_cpu, sz_inp);
        
        Linear_CPU lin_cpu(bs, n_in, n_out);
        Linear_GPU lin_gpu(bs, n_in, n_out);
        set_eq(lin_gpu.weights, lin_cpu.weights, sz_weights);
    
        lin_cpu.forward(inp_cpu, out_cpu);
        lin_gpu.forward(inp_gpu, out_gpu);

        std::cout << "Result of the forward pass" << std::endl; 
        test_res(lin_cpu.out, lin_gpu.out, sz_out);

        lin_cpu.update();
        lin_gpu.update();

        std::cout << "Result of the update" << std::endl;
        std::cout << "Weights" << std::endl; 
        test_res(lin_cpu.weights, lin_gpu.weights, sz_weights);
        std::cout << "Bias" << std::endl;
        test_res(lin_cpu.bias, lin_gpu.bias, n_out);

        lin_cpu.backward();
        lin_gpu.backward();

        std::cout << "Result of the backward pass" << std::endl; 
        test_res(lin_cpu.inp, lin_gpu.inp, sz_inp);
    }

    return 0;
}

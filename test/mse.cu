#include <iostream>

#include "../CPU/mse.h"
#include "../GPU/mse.h"
#include "../utils/utils.h"


int main(){
    int bs;
    float *inp_cpu, *out_cpu, *inp_gpu, *out_gpu;

    for (int i=0; i<8; i++){
        std::cout << "Iteration " << i+1 << std::endl;
        
        bs = random_int(32, 2048);

        inp_cpu = new float[bs];
        hipMallocManaged(&inp_gpu, bs*sizeof(float));

        out_cpu = new float[bs+1];
        hipMallocManaged(&out_gpu, (bs+1)*sizeof(float));
    
        fill_array(inp_cpu, bs);
        set_eq(inp_gpu, inp_cpu, bs);

        fill_array(out_cpu, bs+1);
        set_eq(out_gpu, out_cpu, bs+1);
        
        MSE_CPU mse_cpu(bs);
        MSE_GPU mse_gpu(bs);
    
        mse_cpu.forward(inp_cpu, out_cpu);
        mse_gpu.forward(inp_gpu, out_gpu);
        mse_cpu._forward(inp_cpu, out_cpu);
        mse_gpu._forward(inp_gpu, out_gpu);
    
        std::cout << "Result of the forward pass" << std::endl; 
        std::cout << mse_cpu.out[bs]-mse_gpu.out[bs] << std::endl;
        
        mse_cpu.backward();
        mse_gpu.backward();

        std::cout << "Result of the backward pass" << std::endl; 
        test_res(mse_cpu.inp, mse_gpu.inp, bs);
    }

    return 0;
}

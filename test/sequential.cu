#include <iostream>
#include <vector>

#include "../CPU/linear.h"
#include "../CPU/mse.h"
#include "../CPU/relu.h"
#include "../CPU/sequential.h"
#include "../GPU/linear.h"
#include "../GPU/mse.h"
#include "../GPU/relu.h"
#include "../GPU/sequential.h"
#include "../utils/utils.h"


int main(){
    int bs, n_in, n_out;
    int sz_inp, sz_weights, sz_out;
    float *inp_cpu, *inp_gpu, *out;

    for (int i=0; i<8; i++){
        std::cout << "Iteration " << i+1 << std::endl;
        
        bs = random_int(16, 256);
        n_in = random_int(32, 64);
        n_out = random_int(1, 4);
        
        sz_inp = bs*n_in;
        sz_weights = n_in*n_out;
        sz_out = bs*n_out;

        inp_cpu = new float[sz_inp];
        hipMallocManaged(&inp_gpu, sz_inp*sizeof(float));
    
        fill_array(inp_cpu, sz_inp);
        set_eq(inp_gpu, inp_cpu, sz_inp);

        Linear_CPU* lin_cpu = new Linear_CPU(bs, n_in, n_out);
        Linear_GPU* lin_gpu = new Linear_GPU(bs, n_in, n_out);
        set_eq(lin_gpu->weights, lin_cpu->weights, sz_weights);

        ReLU_CPU* relu_cpu = new ReLU_CPU(sz_out);
        ReLU_GPU* relu_gpu = new ReLU_GPU(sz_out);

        std::vector<Module*> layers_cpu = {lin_cpu, relu_cpu};
        std::vector<Module*> layers_gpu = {lin_gpu, relu_gpu};

        Sequential_CPU seq_cpu(layers_cpu);
        Sequential_GPU seq_gpu(layers_gpu);

        seq_cpu.forward(inp_cpu, out);
        seq_gpu.forward(inp_gpu, out);

        std::cout << "Result of the forward pass" << std::endl; 
        test_res(seq_cpu.layers.back()->out, seq_gpu.layers.back()->out, sz_out);

        seq_cpu.update();
        seq_gpu.update();

        std::cout << "Result of the update" << std::endl;
        std::cout << "Weights" << std::endl; 
        test_res(lin_cpu->weights, lin_gpu->weights, sz_weights);
        std::cout << "Bias" << std::endl;
        test_res(lin_cpu->bias, lin_gpu->bias, n_out);
    }

    return 0;
}

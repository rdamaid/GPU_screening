#include <iostream>

#include "mse.h"
#include "train.h"
#include "../utils/utils.h"

void train_gpu(Sequential_GPU seq, float *inp, float *targ, int bs, int n_in, int n_epochs, int bs_test, float *inp_test, float *targ_test){
    MSE_GPU mse(bs);
    
    int sz_inp = bs*n_in;
    float *cp_inp, *out;
    hipMallocManaged(&cp_inp, sz_inp*sizeof(float));

    for (int i=0; i<n_epochs; i++){
        set_eq(cp_inp, inp, sz_inp);

        seq.forward(cp_inp, out);
        mse.forward(seq.layers.back()->out, targ);
        
        mse.backward();
        seq.update();
    }

    seq.forward(inp, out);
    seq.forward(inp_test, out);

    int tp = 0, tn = 0, fp = 0, fn = 0;

    for (int i=0; i<bs_test; i++){
        float y_hat = seq.layers.back()->out[i];
        if ((y_hat > 0.5) && (targ_test[i] > 0.5)) tp++;        // true positive
        else if ((y_hat < 0.5) && (targ_test[i] < 0.5)) tn++;   // true negative
        else if ((y_hat > 0.5) && (targ_test[i] < 0.5)) fp++;   // false positive
        else if ((y_hat < 0.5) && (targ_test[i] > 0.5)) fn++;   // false negative
        // std::cout << "out " << i << ": "<< seq.layers.back()->out[i] << std::endl;
        // std::cout << "targ " << i << ": "<< targ_test[i] << std::endl;
    }

    std::cout << "TP: "<< tp << std::endl;
    std::cout << "TN: "<< tn << std::endl;
    std::cout << "FP: "<< fp << std::endl;
    std::cout << "FN: "<< fn << std::endl;
    std::cout << "akurasi: "<< (float(tp) + float(tn))/(bs_test) << " %" << std::endl;

    float rec = float(tp)/(float(tp) + float(fn)), prec = float(tp)/(float(tp) + float(fp));
    std::cout << "recall: "<< rec << " %" << std::endl;
    std::cout << "precision: "<< prec << " %" << std::endl;
    std::cout << "f-measure: "<< (2 * prec * rec)/(prec + rec) << " %" << std::endl;

    // mse._forward(seq.layers.back()->out, targ_test);
    // std::cout << "The final loss is: " << targ_test[bs_test] << std::endl;

}

#include "sequential.h"
#include "../utils/utils.h"


void sequential_forward_gpu(float *inp, std::vector<Module*> layers, float *out){
    int sz_out;
    float *curr_out;

    for (int i=0; i<layers.size(); i++){
        Module *layer = layers[i];

        sz_out = layer->sz_out;

        hipMallocManaged(&curr_out, sz_out*sizeof(float));
        layer->forward(inp, curr_out);

        inp = curr_out;
    }

    hipMallocManaged(&curr_out, sizeof(float));
    hipFree(curr_out);
}


void sequetial_update_gpu(std::vector<Module*> layers){
    for (int i=layers.size()-1; 0<=i; i--){
        Module *layer = layers[i];

        layer->update(); 
        layer->backward();
    }
}


Sequential_GPU::Sequential_GPU(std::vector<Module*> _layers){
    layers = _layers;
}


void Sequential_GPU::forward(float *inp, float *out){
    sequential_forward_gpu(inp, layers, out);
}


void Sequential_GPU::update(){
    sequetial_update_gpu(layers);
}

#include <chrono>

#include "linear.h"
#include "relu.h"
#include "sigmoid.h"
#include "train.h"
#include "../data/read_csv.h"


int main(){
    std::chrono::steady_clock::time_point begin, end;

    int bs = 405, n_in = 1281, n_epochs = 100;
    int bs_test = 135;
    int n_hidden1 = 10;
    int n_hidden2 = 3;
    float learning_rate = 0.01f;
    // ReLU

    float *inp, *targ, *inp_test, *targ_test;  
    hipMallocManaged(&inp, bs*n_in*sizeof(float));
    hipMallocManaged(&targ, (bs+1)*sizeof(float));
    hipMallocManaged(&inp_test, bs*n_in*sizeof(float));
    hipMallocManaged(&targ_test, (bs+1)*sizeof(float));
    
    begin = std::chrono::steady_clock::now();
    read_csv(inp,  "/content/drive/MyDrive/Tugas Akhir/kode1/data/nr/X_train.csv");
    read_csv(targ, "/content/drive/MyDrive/Tugas Akhir/kode1/data/nr/y_train.csv");
    read_csv(inp_test,  "/content/drive/MyDrive/Tugas Akhir/kode1/data/nr/X_test.csv");
    read_csv(targ_test, "/content/drive/MyDrive/Tugas Akhir/kode1/data/nr/y_test.csv");
    end = std::chrono::steady_clock::now();
    std::cout << "Data reading time: " << (std::chrono::duration_cast<std::chrono::microseconds>(end-begin).count())/1000000.0f << std::endl;
    
    Linear_GPU* lin1 = new Linear_GPU(bs, n_in, n_hidden1, learning_rate);
    ReLU_GPU* relu1 = new ReLU_GPU(bs*n_hidden1);
    Linear_GPU* lin2 = new Linear_GPU(bs, n_hidden1, 1, learning_rate);

    std::cout << "2" << std::endl;

    std::vector<Module*> layers = {lin1, relu1, lin2};
    Sequential_GPU seq(layers);

    std::cout << "3" << std::endl;

    begin = std::chrono::steady_clock::now();
    train_gpu(seq, inp, targ, bs, n_in, n_epochs, bs_test, inp_test, targ_test);
    end = std::chrono::steady_clock::now();
    std::cout << "Training and Testing time: " << (std::chrono::duration_cast<std::chrono::microseconds>(end-begin).count())/1000000.0f << std::endl;

    return 0;
}

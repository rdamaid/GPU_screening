#include <chrono>

#include "linear.h"
#include "relu.h"
#include "sigmoid.h"
#include "train.h"
#include "../data/read_csv.h"


int main(){
    std::chrono::steady_clock::time_point begin, end;

    int bs = 405, n_in = 1281, n_epochs = 100;
    int bs_test = 135;
    int n_hidden1 = 1000;
    int n_hidden2 = 200;
    int n_hidden3 = 40;
    float learning_rate = 0.001f;
    // ReLU

    float *inp, *targ, *inp_test, *targ_test;  
    hipMallocManaged(&inp, bs*n_in*sizeof(float));
    hipMallocManaged(&targ, (bs+1)*sizeof(float));
    hipMallocManaged(&inp_test, bs*n_in*sizeof(float));
    hipMallocManaged(&targ_test, (bs+1)*sizeof(float));
    
    begin = std::chrono::steady_clock::now();
    read_csv(inp,  "/content/drive/MyDrive/Tugas Akhir/kode1/data/nr/X_train.csv");
    read_csv(targ, "/content/drive/MyDrive/Tugas Akhir/kode1/data/nr/y_train.csv");
    read_csv(inp_test,  "/content/drive/MyDrive/Tugas Akhir/kode1/data/nr/X_test.csv");
    read_csv(targ_test, "/content/drive/MyDrive/Tugas Akhir/kode1/data/nr/y_test.csv");
    end = std::chrono::steady_clock::now();
    std::cout << "Data reading time: " << (std::chrono::duration_cast<std::chrono::microseconds>(end-begin).count())/1000000.0f << std::endl;
    
    Linear_GPU* lin1 = new Linear_GPU(bs, n_in, n_hidden1, learning_rate);
    Sigmoid_GPU* sigmoid1 = new Sigmoid_GPU(bs*n_hidden1);
    Linear_GPU* lin2 = new Linear_GPU(bs, n_hidden1, n_hidden2, learning_rate);
    Sigmoid_GPU* sigmoid2 = new Sigmoid_GPU(bs*n_hidden2);
    Linear_GPU* lin3 = new Linear_GPU(bs, n_hidden2, n_hidden3, learning_rate);
    Sigmoid_GPU* sigmoid3 = new Sigmoid_GPU(bs*n_hidden3);
    Linear_GPU* lin4 = new Linear_GPU(bs, n_hidden3, 1, learning_rate);

    std::vector<Module*> layers = {lin1, sigmoid1, lin2, sigmoid2, lin3, sigmoid3, lin4};
    Sequential_GPU seq(layers);

    begin = std::chrono::steady_clock::now();
    train_gpu(seq, inp, targ, bs, n_in, n_epochs, bs_test, inp_test, targ_test);
    end = std::chrono::steady_clock::now();
    std::cout << "Training and Testing time: " << (std::chrono::duration_cast<std::chrono::microseconds>(end-begin).count())/1000000.0f << std::endl;

    return 0;
}

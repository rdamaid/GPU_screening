#include "hip/hip_runtime.h"
#include "mse.h"


__global__
void mse_forward_gpu(float *inp, float *out, int sz_out){
    int ind = blockDim.x*blockIdx.x + threadIdx.x;

    if (ind < sz_out){
        atomicAdd(&out[sz_out], fdividef(powf(inp[ind]-out[ind], 2), sz_out));
    }
}


__global__
void mse_backward_gpu(float *inp, float *out, int sz_out){
    int ind = blockDim.x*blockIdx.x + threadIdx.x;

    if (ind < sz_out){
        inp[ind] = fdividef(2*(inp[ind]-out[ind]), sz_out);
    }
}


MSE_GPU::MSE_GPU(int _sz_out){
    sz_out = _sz_out;
    
    n_blocks = (sz_out + block_size - 1) / block_size;
}


void MSE_GPU::forward(float *_inp, float *_out){
    inp = _inp;
    out = _out;
}


void MSE_GPU::_forward(float *_inp, float *_out){
    _out[sz_out] = 0.0f;
    
    mse_forward_gpu<<<n_blocks, block_size>>>(_inp, _out, sz_out);
    hipDeviceSynchronize();
}


void MSE_GPU::backward(){
    mse_backward_gpu<<<n_blocks, block_size>>>(inp, out, sz_out);
    hipDeviceSynchronize();
}

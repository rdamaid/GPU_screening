#include "hip/hip_runtime.h"
#include "sigmoid.h"


__global__
void sigmoid_forward_gpu(float *inp, float *out, int sz_out){
    int ind = blockDim.x*blockIdx.x + threadIdx.x;
    
    if (ind < sz_out){
        out[ind] = 1 / (1 + expf(-inp[ind]));
    }
}


__global__
void sigmoid_backward_gpu(float *inp, float *out, int sz_out){
    int ind = blockDim.x*blockIdx.x + threadIdx.x;
    
    if (ind < sz_out){
        inp[ind] = (1 - out[ind]) * out[ind];
    }
}


Sigmoid_GPU::Sigmoid_GPU(int _sz_out){
    sz_out = _sz_out;
    
    n_blocks = (sz_out + block_size - 1) / block_size;
}


void Sigmoid_GPU::forward(float *_inp, float *_out){
    inp = _inp;
    out = _out;

    sigmoid_forward_gpu<<<n_blocks, block_size>>>(inp, out, sz_out);
    hipDeviceSynchronize();
}


void Sigmoid_GPU::backward(){    
    sigmoid_backward_gpu<<<n_blocks, block_size>>>(inp, out, sz_out);
    hipDeviceSynchronize();
}
